#include "hip/hip_runtime.h"
#include "utils.h"
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>


// for details of the formulae, please see https://arxiv.org/pdf/2206.05085.pdf

template <typename scalar_t>
__global__ void prefix_sums_kernel(
    const scalar_t* __restrict__ ws,
    const scalar_t* __restrict__ wts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    scalar_t* __restrict__ ws_inclusive_scan,
    scalar_t* __restrict__ ws_exclusive_scan,
    scalar_t* __restrict__ wts_inclusive_scan,
    scalar_t* __restrict__ wts_exclusive_scan
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    // compute prefix sum of ws and ws*ts
    // [a0, a1, a2, a3, ...] -> [a0, a0+a1, a0+a1+a2, a0+a1+a2+a3, ...]
    thrust::inclusive_scan(thrust::device,
                           ws+start_idx,
                           ws+start_idx+N_samples,
                           ws_inclusive_scan+start_idx);
    thrust::inclusive_scan(thrust::device,
                           wts+start_idx,
                           wts+start_idx+N_samples,
                           wts_inclusive_scan+start_idx);
    // [a0, a1, a2, a3, ...] -> [0, a0, a0+a1, a0+a1+a2, ...]
    thrust::exclusive_scan(thrust::device,
                           ws+start_idx,
                           ws+start_idx+N_samples,
                           ws_exclusive_scan+start_idx);
    thrust::exclusive_scan(thrust::device,
                           wts+start_idx,
                           wts+start_idx+N_samples,
                           wts_exclusive_scan+start_idx);
}


template <typename scalar_t>
__global__ void distortion_loss_fw_kernel(
    const scalar_t* __restrict__ _loss,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> loss
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    loss[ray_idx] = thrust::reduce(thrust::device, 
                                   _loss+start_idx,
                                   _loss+start_idx+N_samples,
                                   (scalar_t)0);
}


std::vector<torch::Tensor> distortion_loss_fw_cu(
    const torch::Tensor ws,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a
){
    const int N_rays = rays_a.size(0), N = ws.size(0);

    auto wts = ws * ts;

    auto ws_inclusive_scan = torch::zeros({N}, ws.options());
    auto ws_exclusive_scan = torch::zeros({N}, ws.options());
    auto wts_inclusive_scan = torch::zeros({N}, ws.options());
    auto wts_exclusive_scan = torch::zeros({N}, ws.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ws.type(), "distortion_loss_fw_cu_prefix_sums", 
    ([&] {
        prefix_sums_kernel<scalar_t><<<blocks, threads>>>(
            ws.data_ptr<scalar_t>(),
            wts.data_ptr<scalar_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            ws_inclusive_scan.data_ptr<scalar_t>(),
            ws_exclusive_scan.data_ptr<scalar_t>(),
            wts_inclusive_scan.data_ptr<scalar_t>(),
            wts_exclusive_scan.data_ptr<scalar_t>()
        );
    }));

    auto _loss = 2*(wts_inclusive_scan*ws_exclusive_scan-
                    ws_inclusive_scan*wts_exclusive_scan) + 1.0f/3*ws*ws*deltas;

    auto loss = torch::zeros({N_rays}, ws.options());

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ws.type(), "distortion_loss_fw_cu", 
    ([&] {
        distortion_loss_fw_kernel<scalar_t><<<blocks, threads>>>(
            _loss.data_ptr<scalar_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            loss.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return {loss, ws_inclusive_scan, wts_inclusive_scan};
}


template <typename scalar_t>
__global__ void distortion_loss_bw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dloss,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws_inclusive_scan,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> wts_inclusive_scan,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dws
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];
    const int end_idx = start_idx+N_samples-1;

    const scalar_t ws_sum = ws_inclusive_scan[end_idx];
    const scalar_t wts_sum = wts_inclusive_scan[end_idx];
    // fill in dL_dws from start_idx to end_idx
    for (int s=start_idx; s<=end_idx; s++){
        dL_dws[s] = dL_dloss[ray_idx] * 2 * (
            (s==start_idx?
                (scalar_t)0:
                (ts[s]*ws_inclusive_scan[s-1]-wts_inclusive_scan[s-1])
            ) + 
            (wts_sum-wts_inclusive_scan[s]-ts[s]*(ws_sum-ws_inclusive_scan[s]))
        );
        dL_dws[s] += dL_dloss[ray_idx] * (scalar_t)2/3*ws[s]*deltas[s];
    }
}


torch::Tensor distortion_loss_bw_cu(
    const torch::Tensor dL_dloss,
    const torch::Tensor ws_inclusive_scan,
    const torch::Tensor wts_inclusive_scan,
    const torch::Tensor ws,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a
){
    const int N_rays = rays_a.size(0), N = ws.size(0);

    auto dL_dws = torch::zeros({N}, dL_dloss.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ws.type(), "distortion_loss_bw_cu", 
    ([&] {
        distortion_loss_bw_kernel<scalar_t><<<blocks, threads>>>(
            dL_dloss.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ws_inclusive_scan.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            wts_inclusive_scan.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            dL_dws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return dL_dws;
}